#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

void initialData(float *ip, int size) {
    for (int i = 0; i < size; i++) {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }
}

__global__ void sumArraysGPU(float *a, float *b, float *c) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    c[i] = a[i] + b[i];
}

int main() {
    int nElem = 1 << 20;
    size_t nBytes = nElem * sizeof(float);

    float *a_h, *b_h, *c_h;
    hipHostMalloc((void **)&a_h, nBytes, hipHostMallocDefault);
    hipHostMalloc((void **)&b_h, nBytes, hipHostMallocDefault);
    hipHostMalloc((void **)&c_h, nBytes, hipHostMallocDefault);

    float *a_d, *b_d, *c_d;
    hipMalloc((void **)&a_d, nBytes);
    hipMalloc((void **)&b_d, nBytes);
    hipMalloc((void **)&c_d, nBytes);

    initialData(a_h, nElem);
    initialData(b_h, nElem);

    hipEvent_t start, stop;
    float ms;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Host to Device
    hipEventRecord(start, 0);
    hipMemcpy(a_d, a_h, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_h, nBytes, hipMemcpyHostToDevice);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    printf("Pinned HtoD memcpy time: %.3f ms\n", ms);

    // Kernel
    dim3 block(1024);
    dim3 grid(nElem / block.x);
    hipEventRecord(start, 0);
    sumArraysGPU<<<grid, block>>>(a_d, b_d, c_d);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    printf("Kernel execution time: %.3f ms\n", ms);

    // Device to Host
    hipEventRecord(start, 0);
    hipMemcpy(c_h, c_d, nBytes, hipMemcpyDeviceToHost);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    printf("Pinned DtoH memcpy time: %.3f ms\n", ms);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    hipHostFree(a_h);
    hipHostFree(b_h);
    hipHostFree(c_h);

    return 0;
}
