#include <hip/hip_runtime.h>
#include <stdio.h>

#define N (1 << 24)  // 16M elements
#define BLOCK_SIZE 256

// Array of Structs
struct ParticleAoS {
    float x, y, z;
};

// Kernel for reading AoS
__global__ void readAoS(ParticleAoS* particles, float* out) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        ParticleAoS p = particles[idx];
        out[idx] = p.x + p.y + p.z;
    }
}

// struct of Arrays 
struct ParticleSoA {
    float* x;
    float* y;
    float* z;
};

// Kernel for reading SoA
__global__ void readSoA(ParticleSoA p, float* out) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        out[idx] = p.x[idx] + p.y[idx] + p.z[idx];
    }
}

void checkCuda(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error (%s): %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

int main() {
    
    ParticleAoS* d_particles_aos;
    float* d_out_aos;
    checkCuda(hipMalloc(&d_particles_aos, sizeof(ParticleAoS) * N), "malloc d_particles_aos");
    checkCuda(hipMalloc(&d_out_aos, sizeof(float) * N), "malloc d_out_aos");

    // This is a coarse way to allocate struct of arrays 
    ParticleSoA p_soa;
    float *d_out_soa;
    checkCuda(hipMalloc(&p_soa.x, sizeof(float) * N), "malloc x");
    checkCuda(hipMalloc(&p_soa.y, sizeof(float) * N), "malloc y");
    checkCuda(hipMalloc(&p_soa.z, sizeof(float) * N), "malloc z");
    checkCuda(hipMalloc(&d_out_soa, sizeof(float) * N), "malloc d_out_soa");

    // Timers
    hipEvent_t start, stop;
    float time_aos, time_soa;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    dim3 block(BLOCK_SIZE);
    dim3 grid((N + block.x - 1) / block.x);

    // Array of Struct timing--expected to be slower 
    hipEventRecord(start);
    readAoS<<<grid, block>>>(d_particles_aos, d_out_aos);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_aos, start, stop);

    // Struct of array timing 
    hipEventRecord(start);
    readSoA<<<grid, block>>>(p_soa, d_out_soa);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_soa, start, stop);

    // 
    printf("Read AoS Time: %.3f ms\n", time_aos);
    printf("Read SoA Time: %.3f ms\n", time_soa);

    // Cleanup
    hipFree(d_particles_aos);
    hipFree(d_out_aos);
    hipFree(p_soa.x);
    hipFree(p_soa.y);
    hipFree(p_soa.z);
    hipFree(d_out_soa);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;

    // Output is here
    // Read AoS Time: 66.859 ms
    // Read SoA Time: 0.879 ms
}
