#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

void initialData(float *ip, int size) {
    for (int i = 0; i < size; i++) {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }
}

__global__ void sumArraysGPU(float *a, float *b, float *c) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    c[i] = a[i] + b[i];
}

int main() {
    int nElem = 1 << 20;
    size_t nBytes = nElem * sizeof(float);

    float *a_h = (float *)malloc(nBytes);
    float *b_h = (float *)malloc(nBytes);
    float *c_h = (float *)malloc(nBytes);

    float *a_d, *b_d, *c_d;
    hipMalloc((void **)&a_d, nBytes);
    hipMalloc((void **)&b_d, nBytes);
    hipMalloc((void **)&c_d, nBytes);

    initialData(a_h, nElem);
    initialData(b_h, nElem);

    // CUDA events for timing
    hipEvent_t start, stop;
    float ms;

    // Host to Device copy
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    hipMemcpy(a_d, a_h, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_h, nBytes, hipMemcpyHostToDevice);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    printf("Pageable HtoD memcpy time: %.3f ms\n", ms);

    // Kernel launch
    dim3 block(1024);
    dim3 grid(nElem / block.x);
    hipEventRecord(start, 0);
    sumArraysGPU<<<grid, block>>>(a_d, b_d, c_d);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    printf("Kernel execution time: %.3f ms\n", ms);

    // Device to Host copy
    hipEventRecord(start, 0);
    hipMemcpy(c_h, c_d, nBytes, hipMemcpyDeviceToHost);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    printf("Pageable DtoH memcpy time: %.3f ms\n", ms);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    free(a_h);
    free(b_h);
    free(c_h);

    return 0;
}
